#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013 Pierre-Andre Cazade
 * Copyright (c) 2013 Florent hedin
 *
 * This file is part of MDBas.
 *
 * MDBas is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MDBas is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MDBas.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdlib.h>
#include <float.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>

#include "global.h"
#include "utils.h"
#include "memory.h"
#include "spme.h"
#include "kernels_utils.h"

#ifdef USING_MPI
#include "parallel.h"
#else
#include "serial.h"
#endif

static int newJob;

static real eEwaldself,systq;

static real *sx,*sy,*sz;

static real *bsp,*qsp;
static real **bsp1,**bsp2,**bsp3;
static real **bsd1,**bsd2,**bsd3;

static cplx *bspc1,*bspc2,*bspc3;
static cplx *epl1,*epl2,*epl3;

static hipfftComplex *d_ftqsp;
static hipfftHandle d_fft3d;

void init_spme(CTRL *ctrl,PARAM *param,PARALLEL *parallel,EWALD *ewald,PBC *box)
{
    int i,m1maxp2,m2maxp2,m3maxp2;

    newJob=1;

    ewald->prec=fmin( fabs( ewald->prec ) , 0.5 );
    ewald->tol=sqrt( fabs( log( ewald->prec * param->cutOff ) ) );

    if(!ctrl->keyAlpha)
        ewald->alpha=sqrt( fabs( log( ewald->prec * param->cutOff * ewald->tol ) ) ) / param->cutOff;

    ewald->tol1=sqrt( -log( ewald->prec * param->cutOff * X2( 2.0 * ewald->tol * ewald->alpha ) ) );

    if(!ctrl->keyMmax)
    {
        ewald->m1max=nint(0.25+box->pa*ewald->alpha*ewald->tol1/PI);
        ewald->m2max=nint(0.25+box->pb*ewald->alpha*ewald->tol1/PI);
        ewald->m3max=nint(0.25+box->pc*ewald->alpha*ewald->tol1/PI);
    }

    m1maxp2=1;
    while( (ewald->m1max>m1maxp2) && (m1maxp2<256) )
    {
        m1maxp2*=2;
    }
    ewald->m1max=2*m1maxp2;

    m2maxp2=1;
    while( (ewald->m2max>m2maxp2) && (m2maxp2<256) )
    {
        m2maxp2*=2;
    }
    ewald->m2max=2*m2maxp2;

    m3maxp2=1;
    while( (ewald->m3max>m3maxp2) && (m3maxp2<256) )
    {
        m3maxp2*=2;
    }
    ewald->m3max=2*m3maxp2;

    epl1=(cplx*)my_malloc(ewald->m1max*sizeof(*epl1));
    epl2=(cplx*)my_malloc(ewald->m2max*sizeof(*epl2));
    epl3=(cplx*)my_malloc(ewald->m3max*sizeof(*epl3));

    bspc1=(cplx*)my_malloc(ewald->m1max*sizeof(*bspc1));
    bspc2=(cplx*)my_malloc(ewald->m2max*sizeof(*bspc2));
    bspc3=(cplx*)my_malloc(ewald->m3max*sizeof(*bspc3));

    sx=(real*)my_malloc(parallel->maxAtProc*sizeof(*sx));
    sy=(real*)my_malloc(parallel->maxAtProc*sizeof(*sy));
    sz=(real*)my_malloc(parallel->maxAtProc*sizeof(*sz));

    bsp=(real*)my_malloc(ewald->nbsp*sizeof(*bsp));

    bsp1=(real**)my_malloc(parallel->maxAtProc*sizeof(*bsp1));
    bsp2=(real**)my_malloc(parallel->maxAtProc*sizeof(*bsp2));
    bsp3=(real**)my_malloc(parallel->maxAtProc*sizeof(*bsp3));

    bsd1=(real**)my_malloc(parallel->maxAtProc*sizeof(*bsd1));
    bsd2=(real**)my_malloc(parallel->maxAtProc*sizeof(*bsd2));
    bsd3=(real**)my_malloc(parallel->maxAtProc*sizeof(*bsd3));

    for(i=0; i<parallel->maxAtProc; i++)
    {
        bsp1[i]=(real*)my_malloc(ewald->nbsp*sizeof(**bsp1));
        bsp2[i]=(real*)my_malloc(ewald->nbsp*sizeof(**bsp2));
        bsp3[i]=(real*)my_malloc(ewald->nbsp*sizeof(**bsp3));

        bsd1[i]=(real*)my_malloc(ewald->nbsp*sizeof(**bsd1));
        bsd2[i]=(real*)my_malloc(ewald->nbsp*sizeof(**bsd2));
        bsd3[i]=(real*)my_malloc(ewald->nbsp*sizeof(**bsd3));
    }

    ewald->mmax=ewald->m1max*ewald->m2max*ewald->m3max;

    qsp=(real*)my_malloc(ewald->mmax*sizeof(*qsp));
    d_ftqsp=(cplx*)hipMalloc(ewald->mmax*sizeof(*d_ftqsp));

}

void spme_free(PARALLEL *parallel)
{
    int i;

    free(epl1);
    free(epl2);
    free(epl3);

    free(bspc1);
    free(bspc2);
    free(bspc3);

    free(sx);
    free(sy);
    free(sz);

    free(bsp);

    for(i=0; i<parallel->maxAtProc; i++)
    {
        free(bsp1[i]);
        free(bsp2[i]);
        free(bsp3[i]);

        free(bsd1[i]);
        free(bsd2[i]);
        free(bsd3[i]);
    }

    free(bsp1);
    free(bsp2);
    free(bsp3);

    free(bsd1);
    free(bsd2);
    free(bsd3);

    free(qsp);

    hipFree(d_ftqsp);
    hipfftDestroy(d_fft3d);

}

void epl_cplx(EWALD *ewald)
{

    int i,hm1max,hm2max,hm3max;

    hm1max=ewald->m1max/2;
    hm2max=ewald->m2max/2;
    hm3max=ewald->m3max/2;

    epl1[0]=cudaComp(1.0);

    for(i=1; i<=hm1max; i++)
    {
        epl1[i]=cudaExpI(TWOPI*(real)i/(real)ewald->m1max);
        epl1[ewald->m1max-i]=cudaConj(epl1[i]);
    }

    epl2[0]=cudaComp(1.0);

    for(i=1; i<=hm2max; i++)
    {
        epl2[i]=cudaExpI(TWOPI*(real)i/(real)ewald->m2max);
        epl2[ewald->m2max-i]=cudaConj(epl2[i]);
    }

    epl3[0]=cudaComp(1.0);

    for(i=1; i<=hm3max; i++)
    {
        epl3[i]=cudaExpI(TWOPI*(real)i/(real)ewald->m3max);
        epl3[ewald->m3max-i]=cudaConj(epl3[i]);
    }

}

void bspcoef(EWALD *ewald)
{

    int i,j,k;
    cplx coeff;

    bsp[0]=0.0;
    bsp[1]=1.0;

    for(k=2; k<ewald->nbsp; k++)
    {

        bsp[k]=0.0;

        for(j=k; j>0; j--)
        {
            bsp[j] = ( (real)j*bsp[j]+(real)(k+1-j)*bsp[j-1] ) / ( (real)k );
        }
    }

    for(i=0; i<ewald->m1max; i++)
    {

        coeff=cudaComp(0.0);

        for(k=0; k<ewald->nbsp-1; k++)
        {
            coeff=cudaAdd(coeff,cudaMul(cudaComp(bsp[k+1]),epl1[( (i*k) % ewald->m1max )]));
        }

        bspc1[i]=cudaDiv(epl1[( ( i* (ewald->nbsp-1) ) % ewald->m1max )],coeff);

    }

    for(i=0; i<ewald->m2max; i++)
    {

        coeff=cudaComp(0.0);

        for(k=0; k<ewald->nbsp-1; k++)
        {
            coeff=cudaAdd(coeff,cudaMul(cudaComp(bsp[k+1]),epl2[( (i*k) % ewald->m2max )]));
        }

        bspc2[i]=cudaDiv(epl2[( ( i* (ewald->nbsp-1) ) % ewald->m2max )],coeff);

    }

    for(i=0; i<ewald->m3max; i++)
    {

        coeff=0.+I*0.0;

        for(k=0; k<ewald->nbsp-1; k++)
        {
            coeff=cudaAdd(coeff,cudaMul(cudaComp(bsp[k+1]),epl3[( (i*k) % ewald->m3max )]));
        }

        bspc3[i]=cudaDiv(epl3[( ( i* (ewald->nbsp-1) ) % ewald->m3max )],coeff);

    }

}

void bspgen(PARALLEL *parallel,EWALD *ewald)
{

    int i,j,k;
    real tsx,tsy,tsz;

    for(i=0; i<parallel->nAtProc; i++)
    {

        bsd1[i][0]=1.0;
        bsd2[i][0]=1.0;
        bsd3[i][0]=1.0;

        bsd1[i][1]=-1.0;
        bsd2[i][1]=-1.0;
        bsd3[i][1]=-1.0;

        bsp1[i][0]=sx[i]-(int)sx[i];
        bsp2[i][0]=sy[i]-(int)sy[i];
        bsp3[i][0]=sz[i]-(int)sz[i];

        bsp1[i][1]=1.0-bsp1[i][0];
        bsp2[i][1]=1.0-bsp2[i][0];
        bsp3[i][1]=1.0-bsp3[i][0];

    }

    for(k=2; k<ewald->nbsp; k++)
    {

        for(i=0; i<parallel->nAtProc; i++)
        {
            bsp1[i][k]=0.0;
            bsp2[i][k]=0.0;
            bsp3[i][k]=0.0;
        }

        for(j=k; j>0; j--)
        {

            if( k == (ewald->nbsp-1) )
            {
                for(i=0; i<parallel->nAtProc; i++)
                {
                    bsd1[i][j]=bsp1[i][j]-bsp1[i][j-1];
                    bsd2[i][j]=bsp2[i][j]-bsp2[i][j-1];
                    bsd3[i][j]=bsp3[i][j]-bsp3[i][j-1];
                }
            }

            for(i=0; i<parallel->nAtProc; i++)
            {
                tsx=sx[i]+(real)j-(int)sx[i];
                tsy=sy[i]+(real)j-(int)sy[i];
                tsz=sz[i]+(real)j-(int)sz[i];

                bsp1[i][j]=(tsx*bsp1[i][j]+((real)(k+1)-tsx)*bsp1[i][j-1])/((real)k);
                bsp2[i][j]=(tsy*bsp2[i][j]+((real)(k+1)-tsy)*bsp2[i][j-1])/((real)k);
                bsp3[i][j]=(tsz*bsp3[i][j]+((real)(k+1)-tsz)*bsp3[i][j-1])/((real)k);
            }

        }

        if( k == (ewald->nbsp-1) )
        {
            for(i=0; i<parallel->nAtProc; i++)
            {
                bsd1[i][0]=bsp1[i][0];
                bsd2[i][0]=bsp2[i][0];
                bsd3[i][0]=bsp3[i][0];
            }
        }

        for(i=0; i<parallel->nAtProc; i++)
        {
            tsx=sx[i]-(int)sx[i];
            tsy=sy[i]-(int)sy[i];
            tsz=sz[i]-(int)sz[i];

            bsp1[i][0]=tsx*bsp1[i][0]/((real)k);
            bsp2[i][0]=tsy*bsp2[i][0]/((real)k);
            bsp3[i][0]=tsz*bsp3[i][0]/((real)k);
        }

    }

}

__global__ void mesh(real d_sx[],real d_x[],real d_y[],real d_z[],
		     real u1,real u2,real u3,int fAtProc,int lAtProc,int mmax)
{
  int ll=blockIdx.x * blockDim.x + threadIdx.x;
  int i=ll+fAtProc;
  
  if(i<lAtProc)
    d_sx[ll]=(real)mmax*(d_x[i]*u1+d_y[i]*u2+d_z[i]*u3+0.5);
}

real spme_energy(PARAM *param,PARALLEL *parallel,EWALD *ewald,PBC *box,const real x[],
                   const real y[],const real z[],real fx[],real fy[],real fz[],
                   const real q[],real stress[6],real *virEwaldRec,real dBuffer[])
{

    int i,ii,j,jj,k,kk,l,ll;
    int m1,m2,m3;
    int hm1max,hm2max,hm3max;

    cplx cam,etmp;

    real vam,qtmp,fact1;
    real tt,bm1,bm2,bm3;
    real rm,rrm,rmx,rmy,rmz;
    real rm1x,rm1y,rm1z,rm2x,rm2y,rm2z;
    real recCutOff,recCutOff2,rAlpha2,rVol;
    real eEwaldRec,eNonNeutral;
    real fbx,fby,fbz;
    real fm[3];
    
    real d_x,d_y,d_z;
    cplx *d_bspc1,*d_bspc2,*d_bspc3;
    cplx *d_bsp1,*d_bsp2,*d_bsp3;
    cplx *d_bsd1,*d_bsd2,*d_bsd3;
    
    dim3 threadsPerBlock(nThrdsX,nThrdsY);
    dim3 nAtBlocks(parallel->nAtProc/threadsPerBlock.x,parallel->nAtProc/threadsPerBlock.y);

    if(newJob)
    {
        newJob=0;

        systq=0.;
        eEwaldself=0.;

        for(i=parallel->fAtProc; i<parallel->lAtProc; i++)
        {
            systq+=q[i];
            eEwaldself+=X2(q[i]);
        }

        if(parallel->nProc>1)
        {
            dBuffer[0]=systq;
            dBuffer[1]=eEwaldself;
            sum_double_para(dBuffer,&(dBuffer[3]),2);
            systq=dBuffer[0];
            eEwaldself=dBuffer[1];
        }

        eEwaldself=-param->chargeConst*ewald->alpha*eEwaldself/SQRTPI;

//     Calculate complex exponentials
        epl_cplx(ewald);

//     Calculate the coefficient of the B-spline
        bspcoef(ewald);
	
# ifdef DOUBLE_CUDA
	hipfftPlan3d(&d_fft3d,ewald->m1max,ewald->m2max,ewald->m3max,HIPFFT_Z2Z);
#else
	hipfftPlan3d(&d_fft3d,ewald->m1max,ewald->m2max,ewald->m3max,HIPFFT_C2C);
#endif

    } //   End if(newJob)

    hm1max=ewald->m1max/2;
    hm2max=ewald->m2max/2;
    hm3max=ewald->m3max/2;

    eEwaldRec=0.;
    *virEwaldRec=0.;

    stress[0]=0.;
    stress[1]=0.;
    stress[2]=0.;
    stress[3]=0.;
    stress[4]=0.;
    stress[5]=0.;

    rVol=TWOPI/box->vol;
    rAlpha2=-0.25/(X2(ewald->alpha));

    //   Set the cutoff in reciprocal space
    recCutOff=fmin( ( (real)ewald->m1max*box->u ) , ( (real)ewald->m2max*box->v ) );
    recCutOff=fmin( recCutOff , (real)ewald->m3max*box->w );
    recCutOff=recCutOff*1.05*TWOPI;
    recCutOff2=X2(recCutOff);
    
//  Memory allocation on the GPU
    
    hipMalloc((void**) &d_x,param->nAtom*sizeof(real));
    hipMalloc((void**) &d_y,param->nAtom*sizeof(real));
    hipMalloc((void**) &d_z,param->nAtom*sizeof(real));
    
    hipMalloc((void**) &d_bspc1,ewald->m1max*sizeof(cplx));
    hipMalloc((void**) &d_bspc2,ewald->m2max*sizeof(cplx));
    hipMalloc((void**) &d_bspc3,ewald->m3max*sizeof(cplx));
    
    hipMalloc((void**) &d_bsp1,parallel->maxAtProc*ewald->nbsp*sizeof(cplx));
    hipMalloc((void**) &d_bsp2,parallel->maxAtProc*ewald->nbsp*sizeof(cplx));
    hipMalloc((void**) &d_bsp3,parallel->maxAtProc*ewald->nbsp*sizeof(cplx));
    
    hipMalloc((void**) &d_bsd1,parallel->maxAtProc*ewald->nbsp*sizeof(cplx));
    hipMalloc((void**) &d_bsd2,parallel->maxAtProc*ewald->nbsp*sizeof(cplx));
    hipMalloc((void**) &d_bsd3,parallel->maxAtProc*ewald->nbsp*sizeof(cplx));
    
// Copy arrays from host to device
  
    hipMemcpy(d_x,x,param->nAtom*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(d_y,y,param->nAtom*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(d_z,z,param->nAtom*sizeof(real),hipMemcpyHostToDevice);
    
    hipMemcpy(d_bspc1,bspc1,param->nAtom*sizeof(cplx),hipMemcpyHostToDevice);
    hipMemcpy(d_bspc2,bspc2,param->nAtom*sizeof(cplx),hipMemcpyHostToDevice);
    hipMemcpy(d_bspc3,bspc3,param->nAtom*sizeof(cplx),hipMemcpyHostToDevice);
    
//   Address atoms to the cells of the mesh [0...mimax]
    
    mesh<<<(parallel->nAtProc/nThrds),nThrds>>>(d_sx,d_x,d_y,d_z,box->u1,box->u2,box->u3,parallel->fAtProc,parallel->lAtProc,ewald->m1max);
    mesh<<<(parallel->nAtProc/nThrds),nThrds>>>(d_sy,d_x,d_y,d_z,box->v1,box->v2,box->v3,parallel->fAtProc,parallel->lAtProc,ewald->m2max);
    mesh<<<(parallel->nAtProc/nThrds),nThrds>>>(d_sz,d_x,d_y,d_z,box->w1,box->w2,box->w3,parallel->fAtProc,parallel->lAtProc,ewald->m3max);

//   Construct the B-splines
    bspgen(parallel,ewald);
    

//   Initialise charge array Q(k1,k2,k3)
    for(i=0; i<ewald->mmax; i++)
        qsp[i]=0.0;

//   Fill charge array Q(k1,k2,k3)
    ll=0;
    for(l=parallel->fAtProc; l<parallel->lAtProc; l++)
    {
        for(i=0; i<ewald->nbsp; i++)
        {
            ii=(int)sx[ll]-i;

            if(ii>=ewald->m1max)
                ii=0;

            if(ii<0)
                ii+=ewald->m1max;

            m1=ewald->m2max*ii;

            for(j=0; j<ewald->nbsp; j++)
            {
                jj=(int)sy[ll]-j;

                if(jj>=ewald->m2max)
                    jj=0;

                if(jj<0)
                    jj+=ewald->m2max;

                m2=ewald->m3max*(jj+m1);

                for(k=0; k<ewald->nbsp; k++)
                {
                    kk=(int)sz[ll]-k;

                    if(kk>=ewald->m3max)
                        kk=0;

                    if(kk<0)
                        kk+=ewald->m3max;

                    m3=kk+m2;
                    qsp[m3]+=q[l]*bsp1[ll][i]*bsp2[ll][j]*bsp3[ll][k];

                }
            }
        }
        ll++;
    }

    if(parallel->nProc>1)
        sum_double_para(qsp,dBuffer,ewald->mmax);

    for(m3=0; m3<ewald->mmax; m3++)
    {
        d_ftqsp[m3]=cudaComp(qsp[m3]);
    }

//   Perform the Fourier Transform of Q(k1,k2,k3)
# ifdef DOUBLE_CUDA
    hipfftExecZ2Z(d_fft3d,d_ftqsp,d_ftqsp,HIPFFT_BACKWARD);
#else
    hipfftExecC2C(d_fft3d,d_ftqsp,d_ftqsp,HIPFFT_BACKWARD);
#endif
    
    for(i=0; i<ewald->m1max; i++)
    {
        ii=i;

        if(i>hm1max)
            ii=i-ewald->m1max;

        m1=ewald->m2max*i;

        tt=TWOPI*(real)ii;

        rm1x=tt*box->u1;
        rm1y=tt*box->u2;
        rm1z=tt*box->u3;

        bm1=creal(bspc1[i]*conj(bspc1[i]));

        for(j=0; j<ewald->m2max; j++)
        {
            jj=j;

            if(j>hm2max)
                jj=j-ewald->m2max;

            m2=ewald->m3max*(j+m1);

            tt=TWOPI*(real)jj;

            rm2x=rm1x+(tt*box->v1);
            rm2y=rm1y+(tt*box->v2);
            rm2z=rm1z+(tt*box->v3);

            bm2=bm1*creal(bspc2[j]*conj(bspc2[j]));

            for(k=0; k<ewald->m3max; k++)
            {

                kk=k;

                if(k>hm3max)
                    kk=k-ewald->m3max;

                m3=k+m2;

                tt=TWOPI*(real)kk;

                rmx=rm2x+(tt*box->w1);
                rmy=rm2y+(tt*box->w2);
                rmz=rm2z+(tt*box->w3);

                bm3=bm2*creal(bspc3[k]*conj(bspc3[k]));

                rm=X2(rmx)+X2(rmy)+X2(rmz);

                if( (rm>DBL_EPSILON) && (rm<=recCutOff2 ) )
                {
                    rrm=1.0/rm;

                    cam=bm3*exp(rAlpha2*rm)*rrm*ftqsp[m3];

                    vam=2.0*(rrm-rAlpha2)*creal(cam*conj(ftqsp[m3]));

                    stress[0]-=vam*rmx*rmx;
                    stress[1]-=vam*rmx*rmy;
                    stress[2]-=vam*rmx*rmz;
                    stress[3]-=vam*rmy*rmy;
                    stress[4]-=vam*rmy*rmz;
                    stress[5]-=vam*rmz*rmz;

                    ftqsp[m3]=cam;

                }
                else
                {
                    ftqsp[m3]=0.0+I*0.0;
                }
            }
        }
    }

    /**   Beginning of the forces calculation section   */

# ifdef DOUBLE_CUDA
    hipfftExecZ2Z(d_fft3d,d_ftqsp,d_ftqsp,HIPFFT_FORWARD);
#else
    hipfftExecC2C(d_fft3d,d_ftqsp,d_ftqsp,HIPFFT_FORWARD);
#endif

    fact1=-2.0*rVol*param->chargeConst;

    ll=0;
    for(l=parallel->fAtProc; l<parallel->lAtProc; l++)
    {
        for(i=0; i<ewald->nbsp; i++)
        {
            ii=(int)sx[ll]-i;

            if(ii>=ewald->m1max)
                ii=0;

            if(ii<0)
                ii+=ewald->m1max;

            m1=ewald->m2max*ii;

            for(j=0; j<ewald->nbsp; j++)
            {
                jj=(int)sy[ll]-j;

                if(jj>=ewald->m2max)
                    jj=0;

                if(jj<0)
                    jj+=ewald->m2max;

                m2=ewald->m3max*(jj+m1);

                for(k=0; k<ewald->nbsp; k++)
                {
                    kk=(int)sz[ll]-k;

                    if(kk>=ewald->m3max)
                        kk=0;

                    if(kk<0)
                        kk+=ewald->m3max;

                    m3=kk+m2;

                    qtmp=creal(ftqsp[m3]);

                    fbx=qtmp*bsd1[ll][i]*bsp2[ll][j]*bsp3[ll][k]*(real)ewald->m1max;
                    fby=qtmp*bsp1[ll][i]*bsd2[ll][j]*bsp3[ll][k]*(real)ewald->m2max;
                    fbz=qtmp*bsp1[ll][i]*bsp2[ll][j]*bsd3[ll][k]*(real)ewald->m3max;

                    fx[l]+=fact1*q[l]*(fbx*box->u1+fby*box->v1+fbz*box->w1);
                    fy[l]+=fact1*q[l]*(fbx*box->u2+fby*box->v2+fbz*box->w2);
                    fz[l]+=fact1*q[l]*(fbx*box->u3+fby*box->v3+fbz*box->w3);

                }
            }
        }
        ll++;
    }

//   Set the sum of the forces to 0

    fm[0]=0.0;
    fm[1]=0.0;
    fm[2]=0.0;

    for(l=parallel->fAtProc; l<parallel->lAtProc; l++)
    {
        fm[0]+=fx[l];
        fm[1]+=fy[l];
        fm[2]+=fz[l];
    }

    if(parallel->nProc>1)
        sum_real_para(fm,dBuffer,3);

    fm[0]/=(real)param->nAtom;
    fm[1]/=(real)param->nAtom;
    fm[2]/=(real)param->nAtom;

    for(l=parallel->fAtProc; l<parallel->lAtProc; l++)
    {
        fx[l]-=fm[0];
        fy[l]-=fm[1];
        fz[l]-=fm[2];
    }

    /**   End of the forces calculation section   */

    etmp=0.0+I*0.0;
    for(m3=0; m3<ewald->mmax; m3++)
    {
        etmp+=ftqsp[m3]*qsp[m3];
    }

    eNonNeutral=-(0.5*PI*param->chargeConst)*(X2(systq/ewald->alpha)/box->vol)/(real)parallel->nProc;

    fact1=rVol*param->chargeConst/(real)parallel->nProc;

    eEwaldRec=creal(etmp);

    stress[0]=fact1*(stress[0]+eEwaldRec)+eNonNeutral;
    stress[1]=fact1*stress[1];
    stress[2]=fact1*stress[2];
    stress[3]=fact1*(stress[3]+eEwaldRec)+eNonNeutral;
    stress[4]=fact1*stress[4];
    stress[5]=fact1*(stress[5]+eEwaldRec)+eNonNeutral;

    eEwaldRec=fact1*eEwaldRec+eNonNeutral+eEwaldself/(real)parallel->nProc;

    *virEwaldRec=-(stress[0]+stress[3]+stress[5]);

    return eEwaldRec;

}

