/*
 * Copyright (c) 2013 Pierre-Andre Cazade
 * Copyright (c) 2013 Florent hedin
 *
 * This file is part of MDBas.
 *
 * MDBas is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MDBas is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MDBas.  If not, see <http://www.gnu.org/licenses/>.
 */

/**
 * \file cuda_utils.c
 * \brief Contains various and general utilitary functions.
 * \author Pierre-Andre Cazade
 * \version alpha-branch
 * \date 2014
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <ctype.h>
#include <float.h>

#include <hip/hip_runtime.h>

#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>

# ifdef DOUBLE_CUDA

__host__ __device__ __inline__ cuDoubleComplex cudaAdd(cuDoubleComplex za,cuDoubleComplex zb)
{
  return cuCadd(za,zb);
}

__host__ __device__ __inline__ cuDoubleComplex cudaSub(cuDoubleComplex za,cuDoubleComplex zb)
{
  return cuCsub(za,zb);
}

__host__ __device__ __inline__ cuDoubleComplex cudaMul(cuDoubleComplex za,cuDoubleComplex zb)
{
  return cuCmul(za,zb);
}

__host__ __device__ __inline__ cuDoubleComplex cudaDiv(cuDoubleComplex za,cuDoubleComplex zb)
{
  return cuCdiv(za,zb);
}

__host__ __device__ __inline__ cuDoubleComplex cudaConj(cuDoubleComplex z)
{
  return cuConj(z);
}

__host__ __device__ __inline__ cuDoubleComplex cudaExpC(cuDoubleComplex z)
{

  cuDoubleComplex res;

  double t = exp(z.x);

  res.x=cos(z.y);
  res.y=sin(z.y);

  res.x *= t;
  res.y *= t;

  return res;

}

__host__ __device__ __inline__ cuDoubleComplex cudaExp(double z)
{

  cuDoubleComplex res;

  res.x=exp(z);
  res.y=0.0;

  return res;

}

__host__ __device__ __inline__ cuDoubleComplex cudaExpI(double z)
{

  cuDoubleComplex res;

  res.x=cos(z);
  res.y=sin(z);

  return res;

}

#else

__host__ __device__ __inline__ hipComplex cudaAdd(hipComplex za,hipComplex zb)
{
  return hipCaddf(za,zb);
}

__host__ __device__ __inline__ hipComplex cudaSub(hipComplex za,hipComplex zb)
{
  return hipCsubf(za,zb);
}

__host__ __device__ __inline__ hipComplex cudaMul(hipComplex za,hipComplex zb)
{
  return hipCmulf(za,zb);
}

__host__ __device__ __inline__ hipComplex cudaDiv(hipComplex za,hipComplex zb)
{
  return hipCdivf(za,zb);
}

__host__ __device__ __inline__ hipComplex cudaConj(hipComplex z)
{
  return hipConjf(z);
}

__host__ __device__ __inline__ hipComplex cudaExpC(hipComplex z)
{

  hipComplex res;

  float t = expf(z.x);

  res.x=cosf(z.y);
  res.y=sinf(z.y);

  res.x *= t;
  res.y *= t;

  return res;

}

__host__ __device__ __inline__ hipComplex cudaExp(float z)
{

  hipComplex res;

  res.x=expf(z);
  res.y=0.f;

  return res;

}

__host__ __device__ __inline__ hipComplex cudaExpI(float z)
{

  hipComplex res;

  res.x=cosf(z);
  res.y=sinf(z);

  return res;

}

#endif