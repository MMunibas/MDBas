#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013 Pierre-Andre Cazade
 * Copyright (c) 2013 Florent hedin
 *
 * This file is part of MDBas.
 *
 * MDBas is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MDBas is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MDBas.  If not, see <http://www.gnu.org/licenses/>.
 */

/**
 * \file cuda_utils.c
 * \brief Contains various and general utilitary functions.
 * \author Pierre-Andre Cazade
 * \version alpha-branch
 * \date 2014
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <ctype.h>
#include <float.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>

#include "global.h"

# ifdef DOUBLE_CUDA

__host__ __device__ __inline__ cplx cudaAdd(cplx za,cplx zb)
{
  return hipCadd(za,zb);
}

__host__ __device__ __inline__ cplx cudaSub(cplx za,cplx zb)
{
  return hipCsub(za,zb);
}

__host__ __device__ __inline__ cplx cudaMul(cplx za,cplx zb)
{
  return hipCmul(za,zb);
}

__host__ __device__ __inline__ cplx cudaDiv(cplx za,cplx zb)
{
  return hipCdiv(za,zb);
}

__host__ __device__ __inline__ cplx cudaConj(cplx z)
{
  return hipConj(z);
}

__host__ __device__ __inline__ cplx cudaExpC(cplx z)
{

  cplx res;

  real t = exp(z.x);

  res.x=cos(z.y);
  res.y=sin(z.y);

  res.x *= t;
  res.y *= t;

  return res;

}

__host__ __device__ __inline__ cplx cudaExp(real z)
{

  cplx res;

  res.x=exp(z);
  res.y=0.0;

  return res;

}

__host__ __device__ __inline__ cplx cudaExpI(real z)
{

  cplx res;

  res.x=cos(z);
  res.y=sin(z);

  return res;

}

__host__ __device__ __inline__ real cudaRe(cplx z)
{
  return z.x;
}

__host__ __device__ __inline__ real cudaIm(cplx z)
{
  return z.y;
}

__host__ __device__ __inline__ real cudaComp(real z)
{
  
  cplx res;
  
  res.x=z;
  res.y=0.0;
  
  return res;
  
}

#else

__host__ __device__ __inline__ cplx cudaAdd(cplx za,cplx zb)
{
  return hipCaddf(za,zb);
}

__host__ __device__ __inline__ cplx cudaSub(cplx za,cplx zb)
{
  return hipCsubf(za,zb);
}

__host__ __device__ __inline__ cplx cudaMul(cplx za,cplx zb)
{
  return hipCmulf(za,zb);
}

__host__ __device__ __inline__ cplx cudaDiv(cplx za,cplx zb)
{
  return hipCdivf(za,zb);
}

__host__ __device__ __inline__ cplx cudaConj(cplx z)
{
  return hipConjf(z);
}

__host__ __device__ __inline__ cplx cudaExpC(cplx z)
{

  cplx res;

  float t = expf(z.x);

  res.x=cosf(z.y);
  res.y=sinf(z.y);

  res.x *= t;
  res.y *= t;

  return res;

}

__host__ __device__ __inline__ cplx cudaExp(float z)
{

  cplx res;

  res.x=expf(z);
  res.y=0.f;

  return res;

}

__host__ __device__ __inline__ cplx cudaExpI(float z)
{

  cplx res;

  res.x=cosf(z);
  res.y=sinf(z);

  return res;

}

__host__ __device__ __inline__ real cudaRe(cplx z)
{
  return z.x;
}

__host__ __device__ __inline__ real cudaIm(cplx z)
{
  return z.y;
}

__host__ __device__ __inline__ real cudaComp(real z)
{
  
  cplx res;
  
  res.x=z;
  res.y=0.0;
  
  return res;
  
}

#endif